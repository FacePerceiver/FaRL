#include "p2i_max.h"

namespace haya_ext {
std::vector<at::Tensor>
p2i_max_forward_gpu(const at::Tensor &points, const at::Tensor &point_features,
                    const at::Tensor &batch_inds, const at::Tensor &background,
                    int kernel_kind, double kernel_radius) {
  return p2i_max_op::forward<gpu_device>(points, point_features, batch_inds,
                                         background, kernel_kind,
                                         kernel_radius);
}

std::vector<at::Tensor> p2i_max_backward_gpu(const at::Tensor &out_grad,
                                             const at::Tensor &out_point_ids,
                                             const at::Tensor &points,
                                             const at::Tensor &point_features,
                                             int kernel_kind,
                                             double kernel_radius) {
  return p2i_max_op::backward<gpu_device>(out_grad, out_point_ids, points,
                                          point_features, kernel_kind,
                                          kernel_radius);
}
} // namespace haya_ext